#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <fstream>
#include <algorithm>

__global__ void findZeroInDegreeNodes(const int *inDegrees, int *flags, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        flags[i] = (inDegrees[i] == 0) ? 1 : 0;
    }
}

__global__ void markProcessedNodes(int *inDegrees, const int *zeroNodes, int count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < count) {
        inDegrees[zeroNodes[i]] = -1;
    }
}

__global__ void reduceInDegree(int *inDegrees, const int *edges, const int *rowPtr, const int *zeroNodes, int count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < count) {
        int node = zeroNodes[i];
        for (int e = rowPtr[node]; e < rowPtr[node+1]; e++) {
            atomicSub(&inDegrees[edges[e]], 1);
        }
    }
}

// Functor to scatter zero-degree nodes
struct ScatterFunctor {
    int *d_flags;
    int *d_positions;
    int *d_zeroNodes;

    __host__ __device__
    ScatterFunctor(int *flags, int *positions, int *zeroNodes)
        : d_flags(flags), d_positions(positions), d_zeroNodes(zeroNodes) {}

    __host__ __device__
    void operator()(int idx) {
        if (d_flags[idx] == 1) {
            int pos = d_positions[idx];
            d_zeroNodes[pos] = idx;
        }
    }
};

int main(int argc, char **argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input>" << std::endl;
        return 1;
    }
    std::string input = argv[1];
    if (input.size() <= 3 || input.substr(input.size() - 3) != ".in") {
        std::cerr << "Error: input file must be a .in file" << std::endl;
        return 1;
    }
    std::string output = argv[2];
    if (output.size() <= 4 || output.substr(input.size() - 3) != ".out") {
        std::cerr << "Error: output file must be a .out file" << std::endl;
        return 1;
    }

    std::ifstream ifs(input);
    if (!ifs) {
        std::cerr << "Error: cannot open file " << input << std::endl;
        return 1;
    }
    std::ofstream ofs(output);
    if (!ofs) {
        ifs.close();
        std::cerr << "Error: cannot open file " << output << std::endl;
        return 1;
    }

    // Start measuring initialization time
    auto start_init = std::chrono::high_resolution_clock::now();

    int n;
    ifs >> n;

    std::vector<int> rowPtrHost(n+1, 0);
    std::vector<int> inDegreeHost(n, 0);
    std::vector<std::vector<int>> adj(n);

    // Reading the graph
    for (int i = 0; i < n; i++) {
        ifs >> inDegreeHost[i];
        for (int j = 0, dep; j < inDegreeHost[i]; j++) {
            ifs >> dep;
            adj[dep].push_back(i);
        }
    }

    for (int i = 0; i < n; i++) {
        rowPtrHost[i+1] = (int)(rowPtrHost[i] + adj[i].size());
    }

    int m = rowPtrHost[n];
    std::vector<int> edgesHost(m);
    for (int i = 0; i < n; i++) {
        std::copy(adj[i].begin(), adj[i].end(), edgesHost.begin() + rowPtrHost[i]);
    }

    int *d_inDegrees, *d_rowPtr, *d_edges;
    hipMalloc(&d_inDegrees, n * sizeof(int));
    hipMalloc(&d_rowPtr, (n+1) * sizeof(int));
    hipMalloc(&d_edges, m * sizeof(int));

    hipMemcpy(d_inDegrees, inDegreeHost.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtr, rowPtrHost.data(), (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, edgesHost.data(), m * sizeof(int), hipMemcpyHostToDevice);

    thrust::device_vector<int> flags(n, 0);
    thrust::device_vector<int> zeroNodes(n);

    int remaining = n;
    const int blockSize = 1024;

    std::vector<std::vector<int>> batches;
    
    // End measuring initialization time
    auto end_init = std::chrono::high_resolution_clock::now();
    auto init_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_init - start_init).count();

    // Start measuring computation time
    auto start_comp = std::chrono::high_resolution_clock::now();

    while (remaining > 0) {
        // Find zero in-degree nodes for this iteration
        int gridSize = (n + blockSize - 1)/ blockSize;
        findZeroInDegreeNodes<<<gridSize, blockSize>>>(d_inDegrees, thrust::raw_pointer_cast(flags.data()), n);
        hipDeviceSynchronize();

        int count = thrust::reduce(flags.begin(), flags.end());
        if (count == 0) {
            // No zero-degree nodes, cycle found
            std::cerr << "Error: cycle found\n";
            return 1;
        }

        // Compute positions via prefix sum for scattering zero-degree nodes
        thrust::device_vector<int> positions(n);
        thrust::exclusive_scan(flags.begin(), flags.end(), positions.begin());

        int *d_flags = thrust::raw_pointer_cast(flags.data());
        int *d_positions = thrust::raw_pointer_cast(positions.data());
        int *d_zeroNodes = thrust::raw_pointer_cast(zeroNodes.data());

        // Scatter indices of zero-degree nodes
        thrust::for_each(thrust::device, thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(n),
                         ScatterFunctor(d_flags, d_positions, d_zeroNodes));

        // Copy the current batch from device to host
        std::vector<int> hostBatch(count);
        thrust::copy_n(zeroNodes.begin(), count, hostBatch.begin());
        batches.push_back(hostBatch);

        // Mark processed nodes
        gridSize = (count + blockSize - 1) / blockSize;
        markProcessedNodes<<<gridSize, blockSize>>>(d_inDegrees, thrust::raw_pointer_cast(zeroNodes.data()), count);
        hipDeviceSynchronize();

        // Reduce in-degree of their neighbors
        gridSize = (count + blockSize - 1) / blockSize;
        reduceInDegree<<<gridSize, blockSize>>>(d_inDegrees, d_edges, d_rowPtr, thrust::raw_pointer_cast(zeroNodes.data()), count);
        hipDeviceSynchronize();

        remaining -= count;

        // Reset flags for next iteration
        thrust::fill(flags.begin(), flags.end(), 0);
    }

    // End measuring computation time
    auto end_comp = std::chrono::high_resolution_clock::now();
    auto comp_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_comp - start_comp).count();

    ofs << batches.size() << "\n";
    for (auto &batch : batches) {
        ofs << batch.size();
        for (int node : batch) {
            ofs << " " << node;
        }
        ofs << "\n";
    }

    hipFree(d_inDegrees);
    hipFree(d_rowPtr);
    hipFree(d_edges);

    // Print initialization and computation times
    std::cout << "Initialization time: " << init_time << " ms" << std::endl;
    std::cout << "Computation time: " << comp_time << " ms" << std::endl;

    return 0;
}