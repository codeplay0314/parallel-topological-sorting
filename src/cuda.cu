#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

struct GlobalConstants {
    int n;
    int pow2n;
} params;

__constant__ GlobalConstants cuParams;

__global__ void kernelCountDependency(int *ifNoDependencyArray, int *dependencyPrefixSum) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= cuParams.n) {
        return;
    }

    ifNoDependencyArray[i] = (dependencyPrefixSum[i * cuParams.pow2n + cuParams.n + 1] == 0);
}

__global__ void kernelXor(int *a, int *b, int size) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        a[i] ^= b[i];
    }
}

__global__ void kernelClearDependencies(int *dependencyMatrix, int *independentIndices, int cnt) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= cnt * cuParams.n) {
        return;
    }

    int i = x % cuParams.n;
    int j = independentIndices[x / cuParams.n];

    dependencyMatrix[i * cuParams.pow2n + j] = 0;
}

/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

/**
* Helper function to print array on device
*/
void print_device_array(int *device_data, size_t length) {
    static int limit = 256;
    printf("[ ");
    if (length <= 2 * limit) {
        for (int i = 0, x; i < length; i++) {
            hipMemcpy(&x, &device_data[i], sizeof(int), hipMemcpyDeviceToHost);
            printf("%d ", x);
        }
    } else {
        for (int i = 0, x; i < limit; i++) {
            hipMemcpy(&x, &device_data[i], sizeof(int), hipMemcpyDeviceToHost);
            printf("%d ", x);
        }
        printf("... ");
        for (int i = 0, x; i < limit; i++) {
            hipMemcpy(&x, &device_data[length - limit + i - 1], sizeof(int), hipMemcpyDeviceToHost);
            printf("%d ", x);
        }
    }
    printf("]\n");
}

__global__ void upsweep_kernel(int *data, int length, int st_scale, int ed_scale) {
    extern __shared__ int sh_data[];

    unsigned int t = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int index = (t * st_scale + st_scale) * 2 - 1;
    unsigned int i = threadIdx.x * 2 + 1;

    sh_data[i - 1] = data[index - st_scale];
    sh_data[i] = data[index];
    __syncthreads();

    for (int scale = st_scale; scale < ed_scale; scale *= 2) {
        if ((t + 1) % (scale / st_scale) == 0) {
            sh_data[i] += sh_data[i - (scale / st_scale)];
        }
        __syncthreads();
    }

    data[index - st_scale] = sh_data[i - 1];
    data[index] = sh_data[i];
}

__global__ void downsweep_kernel(int *data, int length, int st_scale, int ed_scale) {
    extern __shared__ int sh_data[];

    unsigned int t = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int index = (t * ed_scale + ed_scale) * 2 - 1;
    unsigned int i = threadIdx.x * 2 + 1;

    sh_data[i - 1] = data[index - ed_scale];
    sh_data[i] = data[index];
    __syncthreads();

    for (int scale = st_scale / ed_scale / 2; scale >= 1; scale /= 2) {
        if ((t + 1) % scale == 0) {
            int x = sh_data[i - scale];
            sh_data[i - scale] = sh_data[i];
            sh_data[i] += x;
        }
        __syncthreads();
    }

    data[index - ed_scale] = sh_data[i - 1];
    data[index] = sh_data[i];
}

void exclusive_scan(int *device_data, int length) {

    static const int maxThreadNum = 1024;

    length = nextPow2(length);

    std::vector<std::vector<int>> params;

    for (int st_scale = 1; st_scale < length; st_scale *= maxThreadNum * 2) {
        int ed_scale = min(st_scale * maxThreadNum * 2, length);
        int threadsPerBlock = min(maxThreadNum, length / st_scale / 2);
        int numBlocks = max(1, length / (threadsPerBlock * st_scale * 2));
        int sharedMemorySize = threadsPerBlock * 2 * sizeof(int);
        upsweep_kernel<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(device_data, length, st_scale, ed_scale);
        params.push_back(std::vector<int>{st_scale, ed_scale, threadsPerBlock, numBlocks});
        hipDeviceSynchronize();
    }

    int zero = 0;
    hipMemcpy(device_data + length - 1, &zero, sizeof(int), hipMemcpyHostToDevice);

    for (int i = params.size() - 1; i >= 0; i--) {
        int st_scale = params[i][1];
        int ed_scale = params[i][0];
        int threadsPerBlock = params[i][2];
        int numBlocks = params[i][3];
        int sharedMemorySize = threadsPerBlock * 2 * sizeof(int);
        downsweep_kernel<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(device_data, length, st_scale, ed_scale);
        hipDeviceSynchronize();
    }
}

__global__ void kernel_set_output(int *output, int *input, int length) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Set output to 0 if past length
    if (i < length - 1) {
        int val = input[i];
        int above = input[i + 1];

        if (val != above) {
            output[val] = i;
        }
    }
}

int main(const int argc, const char *argv[]) {

    static const size_t maxThreadsPerBlock = 1024;

    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <input>" << std::endl;
        return 1;
    }
    std::string input = argv[1];
    if (input.size() <= 3 || input.substr(input.size() - 3) != ".in") {
        std::cerr << "Error: input file must be a .in file" << std::endl;
        return 1;
    }
    std::string output  = input.substr(0, input.rfind('.') + 1) + "out";

    std::ifstream ifs(input);
    if (!ifs) {
        std::cerr << "Error: cannot open file " << input << std::endl;
        return 1;
    }
    std::ofstream ofs(output);
    if (!ofs) {
        ifs.close();
        std::cerr << "Error: cannot open file " << output << std::endl;
        return 1;
    }
    
    std::vector<std::vector<int>> batches;
    std::vector<int> batch;

    // Read the graph
    ifs >> params.n;
    params.pow2n = nextPow2(params.n + 1);
    hipMemcpyToSymbol(HIP_SYMBOL(cuParams), &params, sizeof(GlobalConstants));

    size_t dependencyMatrixLength = params.pow2n * params.n;
    size_t dependencyMatrixSize = dependencyMatrixLength * sizeof(int);

    std::vector<int> depMatrix(dependencyMatrixLength, 0);
    for (int i = 0, m; i < params.n; ++i) {
        ifs >> m;
        for (int j = 0, dep; j < m; ++j) {
            ifs >> dep;
            depMatrix[i * params.pow2n + dep] = 1;
        }
    }

    int *d_dependencyMatrix;
    hipMalloc(&d_dependencyMatrix, dependencyMatrixSize);
    hipMemcpy(d_dependencyMatrix, depMatrix.data(), dependencyMatrixSize, hipMemcpyHostToDevice);

    int *d_ifNoDependencyArray;
    hipMalloc(&d_ifNoDependencyArray, params.pow2n * sizeof(int));

    int *d_ifSortedArray;
    hipMalloc(&d_ifSortedArray, params.n * sizeof(int));
    hipMemset(d_ifSortedArray, 0, params.n * sizeof(int));

    int *d_dependencyPrefixSum;
    hipMalloc(&d_dependencyPrefixSum, dependencyMatrixSize);

    int *d_independentIndices;
    hipMalloc(&d_independentIndices, (params.n + 1) * sizeof(int));

    int circles_left = params.n;
    while (circles_left > 0) {
        hipMemcpy(d_dependencyPrefixSum, d_dependencyMatrix, dependencyMatrixSize, hipMemcpyDeviceToDevice);
        for (int i = 0; i < params.n; i++) {
            exclusive_scan(d_dependencyPrefixSum + i * params.pow2n, params.n + 1);
        }

        int threadsPerBlock = std::min(maxThreadsPerBlock, static_cast<size_t>(params.n));
        int numBlocks = (params.n + threadsPerBlock - 1) / threadsPerBlock;
        kernelCountDependency<<<numBlocks, threadsPerBlock>>>(d_ifNoDependencyArray, d_dependencyPrefixSum);
        hipDeviceSynchronize();

        // Exclude circles already sorted
        kernelXor<<<numBlocks, threadsPerBlock>>>(d_ifNoDependencyArray, d_ifSortedArray, params.n);
        hipDeviceSynchronize();
        kernelXor<<<numBlocks, threadsPerBlock>>>(d_ifSortedArray, d_ifNoDependencyArray, params.n);
        hipDeviceSynchronize();

        exclusive_scan(d_ifNoDependencyArray, params.n + 1);

        kernel_set_output<<<numBlocks, threadsPerBlock>>>(d_independentIndices, d_ifNoDependencyArray, params.n + 1);

        size_t cnt = 0;
        hipMemcpy(&cnt, d_ifNoDependencyArray + params.n, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        if (cnt == 0) {
            std::cerr << "Error: cycle found" << std::endl;
            return 1;
        }

        batch = std::vector<int>(cnt);
        hipMemcpy(batch.data(), d_independentIndices, cnt * sizeof(int), hipMemcpyDeviceToHost);
        batches.push_back(batch);

        // Clear dependencies
        threadsPerBlock = std::min(maxThreadsPerBlock, cnt * params.n);
        numBlocks = (cnt * params.n + threadsPerBlock - 1) / threadsPerBlock;
        kernelClearDependencies<<<numBlocks, threadsPerBlock>>>(d_dependencyMatrix, d_independentIndices, cnt);
        hipDeviceSynchronize();

        circles_left -= cnt;
    }

    // Write the result
    ofs << batches.size() << std::endl;
    for (const auto &batch : batches) {
        ofs << batch.size();
        for (int i : batch) {
            ofs << ' ' << i;
        }
        ofs << std::endl;
    }

    ifs.close();
    ofs.close();

    // Free allocated memory
    hipFree(d_dependencyMatrix);
    hipFree(d_ifNoDependencyArray);
    hipFree(d_ifSortedArray);
    hipFree(d_dependencyPrefixSum);
    hipFree(d_independentIndices);

    return 0;
}